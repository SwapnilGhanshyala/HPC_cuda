#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#define BLOCKSIZE 1024
// if threa are no syncthreads, there is no need for a warp of the threadblock
// to wait up on other threads of the TBlock to be at the same instruction. So
// first warp that has threads of id 0 to 31 will execute condition 1 and
// condition 2 and then the print statement.
// this will happen due to lack of synchronization between the threads of the
// thread Block. so there is a chance that there will be a "s=1" as output. due
// to __syncthreads, all threads of the block will synchronize whenever the it
// is encountered. so it is guaranteed that warp 0 will not start executing
// condition 2 untill all other warps reach the same point.
// Although the syncthread between condition 1 and cond2 are not needed.
__global__ void dkernel() {
  __shared__ unsigned s;
  if (threadIdx.x == 0)
    s = 0;
  // __syncthreads();
  if (threadIdx.x == 1)
    s += 1;
  // __synchreads();
  if (threadIdx.x == 100)
    s += 2;
  // __syncthreads();
  if (threadIdx.x == 0)
    printf("s=%d,", s);
}
int main() {
  for (int i = 0; i < 36000; i++) {
    dkernel<<<2, BLOCKSIZE>>>();
    hipDeviceSynchronize();
  }
  return 0;
}