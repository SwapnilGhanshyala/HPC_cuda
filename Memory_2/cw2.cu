// there is a link list on the cpu, and it has a next pointer
// copy the ll from CPU to GPU
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>
struct Node {
  int id;
  int val;
  Node *next;
};
__global__ void passLL(Node *arr) {}

int main() {
  Node arr[10];
  Node *darr;
  hipMalloc(&darr, 10 * sizeof(Node));
  hipMemcpy(darr, arr, 10, hipMemcpyHostToDevice);

  return 0;
}