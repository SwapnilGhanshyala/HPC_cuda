
#include <hip/hip_runtime.h>
#include <cmath>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>

#define N 4000
struct Point {
  int x, y;
} arr[N];

__global__ void calcAVG(unsigned *darrx, unsigned *darry, float *globalAVG,
                        unsigned *globalSum, unsigned size) {
  unsigned threadId = blockIdx.x * blockDim.x + threadIdx.x;
  int first = threadId * 4, last = threadId * 4 + 3;
  unsigned sum = 0;
  for (int i = first; i < size && i <= last; i++) {
    sum += darrx[i];
  }
  atomicAdd(globalAVG, (float)sum / size);
  __syncthreads(); // it needs synchronization across TBlocks and not
                   // __syncthreads
  bool flag = false;
  for (int i = first; i < size && i <= last; i++)
    if (darry[i] > *globalAVG) {
      flag = true;
      break;
    }
  if (flag)
    for (int i = first; i < size && i <= last; i++)
      darry[i] = *globalAVG;
  else
    for (int i = first; i < size && i <= last; i++)
      (*globalSum) += darry[i];
}
int main() {
  int arrx[N];
  int arry[N];
  for (int i = 0; i < N; i++) {
    arrx[i] = arr[i].x = rand() % 10;
    arry[i] = arr[i].y = rand() % 10;
  }

  unsigned *darrx, *darry, *globalSum;
  float *globalAVG;
  hipMalloc(&darrx, sizeof(int) * N);
  hipMalloc(&darry, sizeof(int) * N);
  hipMalloc(&globalAVG, sizeof(float));
  hipMalloc(&globalSum, sizeof(int));
  hipMemcpy(darrx, arrx, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(darry, arry, N * sizeof(int), hipMemcpyHostToDevice);
  unsigned gridSize =
      static_cast<unsigned>(ceil(static_cast<double>(N) / 128.0));
  calcAVG<<<gridSize, 32>>>(darrx, darry, globalAVG, globalSum, N);
  hipDeviceSynchronize();
  unsigned *rarry, *rglobalSum;
  float *rglobalAVG;
  hipMemcpy(rarry, darry, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(rglobalAVG, globalAVG, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(rglobalSum, globalSum, sizeof(int), hipMemcpyDeviceToHost);
  printf("globalAVG is %f\n", *rglobalAVG);
  printf("globalSum is %u \n", *rglobalSum);

  return 0;
}