#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <time.h>

__global__ void squaregpu1(unsigned *matrix, unsigned *result) {
  unsigned ii = blockDim.x * blockIdx.x;
  unsigned j = threadIdx.x;
  unsigned matrixsize = blockDim.x;
  for (unsigned k = 0; k < matrixsize; k++)
    result[ii + j] += matrix[ii + k] * matrix[k * blockDim.x + j];
}
// __global__ void squaregpu1(unsigned *matrix, unsigned *result) {
//   unsigned row = blockIdx.x;
//   unsigned col = threadIdx.x;
//   unsigned matrixsize = blockDim.x;
//   unsigned sum = 0;
//   for (unsigned k = 0; k < matrixsize; k++) {
//     sum += matrix[row * matrixsize + k] * matrix[k * matrixsize + col];
//   }
//   result[row * matrixsize + col] = sum;
// }
void squarecpu(unsigned *matrix, unsigned *result, unsigned matrixsize) {
  for (unsigned ii = 0; ii < matrixsize; ii++) {
    for (unsigned jj = 0; jj < matrixsize; jj++) {
      for (unsigned kk = 0; kk < matrixsize; ++kk) {
        result[ii * matrixsize + jj] +=
            matrix[ii * matrixsize + kk] * matrix[kk * matrixsize + jj];
      }
    }
  }
}
int main() {
  unsigned matrix[4096];
  unsigned resultcpu[4096] = {0};
  unsigned resultgpu[4096] = {0};
  unsigned *dresultgpu, *dmatrix;
  for (int i = 0; i < 4096; i++) {
    matrix[i] = rand() % 10;
    // resultcpu[i] = 0;
    // resultgpu[i] = 0;
  }
  unsigned matrixsize = 64;

  unsigned start = clock();
  squarecpu(matrix, resultcpu, matrixsize);
  unsigned end = clock();
  double cpu_time_used = (((double)(end - start)) / CLOCKS_PER_SEC) * 1000;
  printf("time taken by cpu is %f ms\n", cpu_time_used);

  unsigned startgpu = clock();
  hipSetDevice(1);
  hipMalloc(&dresultgpu, 4096 * sizeof(int));
  hipMalloc(&dmatrix, 4096 * sizeof(int));
  hipMemcpy(dresultgpu, resultgpu, 4096 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dmatrix, matrix, 4096 * sizeof(int), hipMemcpyHostToDevice);
  squaregpu1<<<matrixsize, matrixsize>>>(dmatrix, dresultgpu);
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(err));
    return -1;
  }
  hipMemcpy(resultgpu, dresultgpu, 4096 * sizeof(int), hipMemcpyDeviceToHost);
  unsigned endgpu = clock();
  double gpu_time_used =
      (((double)(endgpu - startgpu)) / CLOCKS_PER_SEC) * 1000;
  printf("time taken by gpu is %f ms\n", gpu_time_used);

  // CHECK result
  for (int i = 0; i < 4096; i++) {
    if (resultcpu[i] != resultgpu[i]) {
      printf("Incorrect result at index %d: CPU %u, GPU %u\n", i, resultcpu[i],
             resultgpu[i]);
      break;
    }
  }
  return 0;
}
