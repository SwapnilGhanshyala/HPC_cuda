#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#define BLOCKSIZE 34
// if BLOCKSIZE is < warp size, then there is a automatic synchronization
// between warps(single in this case) in the threadBlock, therefore no need for
// __syncthreads. syncthreads is needed is size is greater.
__global__ void dkernel() {
  __shared__ char str[BLOCKSIZE + 1];
  str[threadIdx.x] = 'A' + (threadIdx.x + blockIdx.x) % BLOCKSIZE;
  if (threadIdx.x == 0) {
    str[BLOCKSIZE] = '\0';
  }
  __syncthreads();
  // without syncthreads, there is no guarantee that other warps of the thread
  // block have finished writing to the shared location
  if (threadIdx.x == 0) {
    printf("%d:%s\n", blockIdx.x, str);
  }
}
int main() {
  dkernel<<<10, BLOCKSIZE>>>();
  hipDeviceSynchronize();
}