#include "hip/hip_runtime.h"
// write a kernel to vary the degree of coalesing from 1 to 32 based on an input
// argument. basically n is the stride
__global__ void varyDOC(unsigned *a, int n) {
  a[threadIdx.x * n] // I think this works
                     // if n=1 then consequtive and DOC is 32
  // if n==2 then 32 threads will access alternate location over 64 memory
  // locations
}