#include "hip/hip_runtime.h"
// M is 1024 x 1024
// each Tblock works on 1024 elements that is 1 row

#include <stdio.h>
__global__ void replaceNoSharedMemory(unsigned *M) {
  unsigned jj = threadIdx.x;
  if (jj < 1023) {
    unsigned ii = blockIdx.x * 1024;
    // M[ii + jj] = M[ii + jj] + M[ii + jj + 1];
    atomicAdd(&M[ii + jj], M[ii + jj] + M[ii + jj + 1]);
  }
}
__global__ void replaceWithSharedMemory(unsigned *M) {
  // STEP 1: copy to shared memory
  // STEP 2: update the shared memory
  // STEP 3: copy back to global memory
  // even in this case a barrier/ or synchronization of some sort is needed.
  // to make sure that all the threads have completed the call.
  // to avoid using a barrier here, in step 2 , read from global memory and
  // write to shared memory.
  // but still step 2 and step 3 need a barrier because other warps might be
  // reading from global memory (step 2) when this warp is writing to global
  // memory(step 3).

  __shared__ unsigned nmij;
  unsigned jj = threadIdx.x;
  if (jj < 1023) {
    unsigned ii = blockIdx.x * 1024;
    nmij = M[ii + jj] + M[ii + jj + 1];
    __syncthreads();
    M[ii + jj] = nmij;
  }
}

int main() {
  const int size = 1024;
  const int bytes = size * sizeof(unsigned);

  // Allocate host memory
  unsigned h_input[size], h_output[size];

  // Initialize input data
  for (int i = 0; i < size; ++i) {
    h_input[i] = static_cast<float>(i);
  }

  // Allocate device memory
  unsigned *d_input;
  hipMalloc(&d_input, bytes);

  // Copy input data to device
  hipMemcpy(d_input, h_input, bytes, hipMemcpyHostToDevice);

  // Define block size and grid size
  const int blockSize = 256;
  const int gridSize = (size + blockSize - 1) / blockSize;

  // Launch kernel with dynamic shared memory allocation
  replaceWithSharedMemory<<<1024, 1024>>>(d_input);

  // Copy output data back to host
  hipMemcpy(h_output, d_input, bytes, hipMemcpyDeviceToHost);

  // Print some output data
  for (int i = 0; i < 10; ++i) {
    std::cout << h_output[i] << " ";
  }
  std::cout << std::endl;

  // Free device memory
  hipFree(d_input);
  hipFree(d_output);

  return 0;
}