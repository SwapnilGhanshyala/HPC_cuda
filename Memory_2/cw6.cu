
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

__global__ void dynshared() {
  extern __shared__ int s[];
  s[threadIdx.x] = threadIdx.x;
  __syncthreads();
  if (threadIdx.x % 2)
    printf("%d\n", s[threadIdx.x]);
}

int main() {
  int n;
  scanf("%d", &n);
  dynshared<<<1, n, n * sizeof(int)>>>();
  hipDeviceSynchronize();
}