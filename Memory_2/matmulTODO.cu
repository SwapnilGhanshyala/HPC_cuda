#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

__global__ void CoalesingAwareSquaregpu(unsigned *matrix, unsigned *result) {
  unsigned ii = blockDim.x * blockIdx.x;
  unsigned k = threadIdx.x;
  unsigned matrixsize = blockDim.x;
  for (unsigned j = 0; j < matrixsize; j++)
    // result[ii + j] += matrix[ii + k] * matrix[k * blockDim.x + j];
    atomicAdd(&result[ii + j], matrix[ii + k] * matrix[k * matrixsize + j]);
}

__global__ void squaregpu(unsigned *matrix, unsigned *result) {
  unsigned ii = blockDim.x * blockIdx.x;
  unsigned j = threadIdx.x;
  unsigned matrixsize = blockDim.x;
  for (unsigned k = 0; k < matrixsize; k++)
    result[ii + j] += matrix[ii + k] * matrix[k * blockDim.x + j];
}
void squarecpu(unsigned *matrix, unsigned *result, unsigned matrixsize) {
  for (unsigned ii = 0; ii < matrixsize; ii++) {
    for (unsigned jj = 0; jj < matrixsize; jj++) {
      for (unsigned kk = 0; kk < matrixsize; ++kk) {
        result[ii * matrixsize + jj] +=
            matrix[ii * matrixsize + kk] * matrix[kk * matrixsize + jj];
      }
    }
  }
}

void cacheAwareSquarecpu(unsigned *matrix, unsigned *result,
                         unsigned matrixsize) {
  for (unsigned ii = 0; ii < matrixsize; ii++) {
    for (unsigned kk = 0; kk < matrixsize; ++kk) {
      for (unsigned jj = 0; jj < matrixsize; jj++) {
        result[ii * matrixsize + jj] +=
            matrix[ii * matrixsize + kk] * matrix[kk * matrixsize + jj];
      }
    }
  }
}
int main() {
  unsigned matrix[4096];
  unsigned resultcpu[4096] = {0};
  unsigned resultgpu[4096] = {0};
  unsigned *dresultgpu, *dmatrix;
  for (int i = 0; i < 4096; i++) {
    matrix[i] = rand() % 10;
  }
  unsigned matrixsize = 64;

  unsigned start = clock();
  // squarecpu(matrix, resultcpu, matrixsize);
  cacheAwareSquarecpu(matrix, resultcpu, matrixsize);
  unsigned end = clock();
  double cpu_time_used = (((double)(end - start)) / CLOCKS_PER_SEC) * 1000;
  printf("time taken by cpu is %f ms\n", cpu_time_used);

  hipSetDevice(1);
  hipMalloc(&dresultgpu, 4096 * sizeof(int));
  hipMalloc(&dmatrix, 4096 * sizeof(int));
  unsigned startgpu = clock();
  hipMemcpy(dresultgpu, resultgpu, 4096 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dmatrix, matrix, 4096 * sizeof(int), hipMemcpyHostToDevice);
  // squaregpu<<<matrixsize, matrixsize>>>(dmatrix, dresultgpu);
  CoalesingAwareSquaregpu<<<matrixsize, matrixsize>>>(dmatrix, dresultgpu);
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(err));
    return -1;
  }
  hipMemcpy(resultgpu, dresultgpu, 4096 * sizeof(int), hipMemcpyDeviceToHost);
  unsigned endgpu = clock();
  double gpu_time_used =
      (((double)(endgpu - startgpu)) / CLOCKS_PER_SEC) * 1000;
  printf("time taken by gpu is %f ms\n", gpu_time_used);

  // CHECK result
  for (int i = 0; i < 4096; i++) {
    if (resultcpu[i] != resultgpu[i]) {
      printf("Incorrect result at index %d: CPU %u, GPU %u\n", i, resultcpu[i],
             resultgpu[i]);
      break;
    }
  }
  return 0;
}