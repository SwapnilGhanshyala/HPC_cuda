#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  int device;
  hipGetDevice(&device);
  std::cout << "Default CUDA device is device " << device << std::endl;

  return 0;
}
