#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printDetails() {
  if (blockIdx.x == 63)
    printf("GridDim : %d, BlockIdx : %d, ThreadIdx.x : %d\n", gridDim.x,
           blockIdx.x, threadIdx.x);
}

int main() {
  hipSetDevice(1);
  printDetails<<<64, 72>>>();
  hipDeviceSynchronize();
  return 0;
}