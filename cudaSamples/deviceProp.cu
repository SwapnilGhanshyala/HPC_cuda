#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  int device = 0; // You can change this to the device index you want to query
  hipDeviceProp_t deviceProp;

  hipError_t error_id = hipGetDeviceProperties(&deviceProp, device);
  if (error_id != hipSuccess) {
    std::cerr << "hipGetDeviceProperties failed! Error: "
              << hipGetErrorString(error_id) << std::endl;
    return EXIT_FAILURE;
  }

  std::cout << "Warp size: " << deviceProp.warpSize << std::endl;

  return EXIT_SUCCESS;
}
