#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);

  if (error_id != hipSuccess) {
    std::cout << "hipGetDeviceCount returned " << static_cast<int>(error_id)
              << " -> " << hipGetErrorString(error_id) << std::endl;
    std::cout << "Result = FAIL" << std::endl;
    return EXIT_FAILURE;
  }

  std::cout << "There are " << deviceCount
            << " CUDA capable devices on your system." << std::endl;
  return EXIT_SUCCESS;
}
