
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>

__global__ void K(int *x) {
  *x = 0;
  printf("x = %d\n", *x);
}
int main() {
  int *x;
  K<<<2, 10>>>(x);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  printf("error=%d, %s, %s\n", err, hipGetErrorName(err),
         hipGetErrorString(err));
  return 0;
}