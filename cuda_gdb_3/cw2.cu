
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void K(int *p) {
  *p = 0;
  printf("%d\n", *p);
}

int main() {
  int *x, *y;
  hipMalloc(&x, sizeof(int));
  K<<<2, 10>>>(x);
  y = x;
  hipFree(y);
  K<<<2, 10>>>(x);
  hipDeviceSynchronize();
  return 0;
}