
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void removeDiv(int x, int y, int z) {
  // assert(x==y || x==z);
  if (x == y)
    x = z;
  else
    x = y;
  // 1. we can remove the else part
  // can we predicate it?
  bool cond = x == y;
  x = cond * z + (1 - cond) * y;
  // OR
  x = (x == y) ? z : y;
}