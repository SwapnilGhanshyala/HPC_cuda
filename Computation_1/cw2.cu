
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdio.h>

#define N 100

__global__ void cw2(int *d_a) {
  if (threadIdx.x < N)
    d_a[threadIdx.x] = threadIdx.x * threadIdx.x;
}
int main() {
  int a[N], *d_a;
  hipMalloc(&d_a, N * sizeof(int));
  cw2<<<1, N>>>(d_a);
  hipDeviceSynchronize(); // optional
  hipMemcpy(a, d_a, N * sizeof(int),
             hipMemcpyDeviceToHost); // O(N) operation theoretically
  // Opt 1: it can get data in blocks instead of element by element, since block
  // size is constant then still O(N)
  // Opt 2: latency hiding using cudaMemcpyAsync
  printf("Printing on host\n");
  for (int i = 0; i < N; i++)
    printf("%d\n", a[i]);
  return 0;
}