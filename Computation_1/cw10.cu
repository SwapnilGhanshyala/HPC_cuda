
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
// code for dod = 4
__global__ void dod4(unsigned *vector, unsigned vectorsize) {
  unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < vectorsize)
    switch (id % 4) {
    case 0:
      vector[id] = 0;
      break;
    case 1:
      vector[id] = 1;
      break;
    case 2:
      vector[id] = 2;
      break;
    case 3:
      vector[id] = 3;
      break;
    }
}

int main() {
  unsigned resultgpu[4096];
  unsigned *dresultgpu;
  for (int i = 0; i < 4096; i++) {
    resultgpu[i] = 0;
  }
  unsigned startgpu = clock();

  hipMalloc(&dresultgpu, 4096 * sizeof(int));
  hipMemcpy(dresultgpu, resultgpu, 4096 * sizeof(int), hipMemcpyHostToDevice);

  dod4<<<64, 64>>>(dresultgpu, 4096);
  hipMemcpy(resultgpu, dresultgpu, 4096 * sizeof(int), hipMemcpyDeviceToHost);
  unsigned endgpu = clock();

  double gpu_time_used =
      (((double)(endgpu - startgpu)) / CLOCKS_PER_SEC) * 1000;
  printf("time taken by gpu is %f ms\n", gpu_time_used);
  for (int i = 0; i < 4096; i++)
    printf("%d, ", resultgpu[i]);
  printf("\n");
  return 0;
}