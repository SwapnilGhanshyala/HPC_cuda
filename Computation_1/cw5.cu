
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 5
#define M 6

__global__ void init2DArray(unsigned *arr) {
  int id = threadIdx.x * blockDim.y + threadIdx.y;
  arr[id] = id;
}

int main() {
  dim3 block(N, M, 1);
  unsigned arr[N * M], *dmatrix;
  hipMalloc(&dmatrix, N * M * sizeof(unsigned));
  init2DArray<<<1, block>>>(dmatrix);
  hipMemcpy(arr, dmatrix, N * M * sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < N * M; i++)
    printf("%d , ", arr[i]);
  printf("\n");
  return 0;
}