#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <time.h>

__global__ void squaregpu1(unsigned *matrix, unsigned *result) {
  unsigned row = blockIdx.x;
  unsigned col = threadIdx.x;
  unsigned matrixsize = gridDim.x; // Assuming square matrix
  unsigned sum = 0;
  for (unsigned k = 0; k < matrixsize; k++) {
    sum += matrix[row * matrixsize + k] * matrix[k * matrixsize + col];
  }
  result[row * matrixsize + col] = sum;
}

void squarecpu(unsigned *matrix, unsigned *result, unsigned matrixsize) {
  for (unsigned ii = 0; ii < matrixsize; ii++) {
    for (unsigned jj = 0; jj < matrixsize; jj++) {
      for (unsigned kk = 0; kk < matrixsize; ++kk) {
        result[ii * matrixsize + jj] +=
            matrix[ii * matrixsize + kk] * matrix[kk * matrixsize + jj];
      }
    }
  }
}

int main() {
  unsigned matrix[4096];
  unsigned resultcpu[4096] = {0}; // Initialize to 0
  unsigned resultgpu[4096] = {0}; // Initialize to 0
  unsigned *dresultgpu, *dmatrix;

  // Initialize matrix with random values
  for (int i = 0; i < 4096; i++) {
    matrix[i] = rand() % 10;
  }

  unsigned matrixsize = 64;

  // Measure CPU time
  unsigned start = clock();
  squarecpu(matrix, resultcpu, matrixsize);
  unsigned end = clock();
  double cpu_time_used = (((double)(end - start)) / CLOCKS_PER_SEC) * 1000;
  printf("time taken by cpu is %f ms\n", cpu_time_used);

  // Allocate memory on the device and initialize
  hipMalloc(&dresultgpu, 4096 * sizeof(unsigned));
  hipMalloc(&dmatrix, 4096 * sizeof(unsigned));

  // Initialize GPU memory to 0
  hipMemset(dresultgpu, 0, 4096 * sizeof(unsigned));

  // Copy data from host to device
  hipMemcpy(dmatrix, matrix, 4096 * sizeof(unsigned), hipMemcpyHostToDevice);

  // Measure GPU time
  unsigned startgpu = clock();
  squaregpu1<<<matrixsize, matrixsize>>>(dmatrix, dresultgpu);
  // Synchronize and check for errors
  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(err));
    return -1;
  }
  unsigned endgpu = clock();
  double gpu_time_used =
      (((double)(endgpu - startgpu)) / CLOCKS_PER_SEC) * 1000;

  // Copy result back to host
  hipMemcpy(resultgpu, dresultgpu, 4096 * sizeof(unsigned),
             hipMemcpyDeviceToHost);

  // Print GPU time
  printf("time taken by gpu is %f ms\n", gpu_time_used);

  // CHECK result
  for (int i = 0; i < 4096; i++) {
    if (resultcpu[i] != resultgpu[i]) {
      printf("Incorrect result at index %d: CPU %u, GPU %u\n", i, resultcpu[i],
             resultgpu[i]);
      break;
    }
  }

  // Free device memory
  hipFree(dresultgpu);
  hipFree(dmatrix);

  return 0;
}
