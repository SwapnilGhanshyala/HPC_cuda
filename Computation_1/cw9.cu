
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>
#include <stdio.h>
#include <time.h>
__global__ void warpCondition(unsigned *vector, unsigned vectorsize) {
  unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id % 2)
    vector[id] = id;
  else
    vector[id] = vectorsize * vectorsize;
  vector[id]++;
}
int main() {
  unsigned resultgpu[4096];
  unsigned *dresultgpu;
  for (int i = 0; i < 4096; i++) {
    resultgpu[i] = 0;
  }
  unsigned startgpu = clock();

  hipMalloc(&dresultgpu, 4096 * sizeof(int));
  hipMemcpy(dresultgpu, resultgpu, 4096 * sizeof(int), hipMemcpyHostToDevice);

  warpCondition<<<64, 64>>>(dresultgpu, 4096);
  hipMemcpy(resultgpu, dresultgpu, 4096 * sizeof(int), hipMemcpyDeviceToHost);
  unsigned endgpu = clock();

  double gpu_time_used =
      (((double)(endgpu - startgpu)) / CLOCKS_PER_SEC) * 1000;
  printf("time taken by gpu is %f ms\n", gpu_time_used);
  for (int i = 0; i < 4096; i++)
    printf("%d, ", resultgpu[i]);
  printf("\n");
  return 0;
}