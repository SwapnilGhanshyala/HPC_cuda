
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void initArray(int *arr, int len) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < len)
    arr[id] = 0;
}

__global__ void addId(int *arr, int len) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < len)
    arr[id] += id;
}

int main() {
  int N = 8000;
  int arr[N], *da;
  hipMalloc(&da, N * sizeof(int));

  //  Below will not work for N=8000 because blockSize is limited to 1024 on
  //  this pc
  //   initArray<<<1, N>>>(da, N);
  //   cudaDeviceSynchronize();
  //   addId<<<1, N>>>(da, N);
  //   cudaDeviceSynchronize();

  initArray<<<10, N / 10>>>(da, N);
  hipDeviceSynchronize(); // optional between 2 kernel launches.
  addId<<<10, N / 10>>>(da, N);
  hipDeviceSynchronize();

  hipMemcpy(arr, da, N * sizeof(int), hipMemcpyDeviceToHost);
  for (int i = 0; i < N; i++)
    printf("Id %d : %d\n ", i, arr[i]);
}