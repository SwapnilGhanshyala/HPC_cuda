#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 100
__global__ void cw1()
{
    printf("%d\n",threadIdx.x*threadIdx.x);
}
int main(){
    cw1<<<1,N>>>();
    hipDeviceSynchronize();
    return 0;
}