// Large amount of data
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <math.h>
#include <stdio.h>
__global__ void init2DArray(unsigned *m, unsigned len) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < len)
    m[id] = id;
}
#define BLOCKSIZE 1024
int main(int nn, char *str[]) {
  unsigned N = atoi(str[1]);
  unsigned *vector, *hvector;
  hipMalloc(&vector, N * sizeof(unsigned));
  hvector = (unsigned *)malloc(N * sizeof(unsigned));

  unsigned nblocks = ceil(
      (float)N / BLOCKSIZE); // use floating point division not integer division
  printf("nblocks = %d\n", nblocks);
  init2DArray<<<nblocks, BLOCKSIZE>>>(vector, N);
  hipMemcpy(hvector, vector, N * sizeof(int), hipMemcpyDeviceToHost);
  for (unsigned i = 0; i < N; i++)
    printf("%d , ", hvector[i]);
  return 0;
}