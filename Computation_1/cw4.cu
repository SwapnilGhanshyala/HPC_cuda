// #include <__clang_cuda_builtin_vars.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
// __global__ int N = 0;
__global__ void dkernel() {
  if (threadIdx.x == 0 && blockIdx.x == 0 && threadIdx.y == 0 &&
      blockIdx.y == 0 && threadIdx.z == 0 && blockIdx.z == 0)
    printf("%d, %d, %d, %d, %d, %d \n", gridDim.x, gridDim.y, gridDim.z,
           blockDim.x, blockDim.y, blockDim.z);
}
int main() {
  //   int N = 8000;
  dim3 grid(2, 5, 1);
  dim3 block(8, 10, 10);
  dkernel<<<grid, block>>>();
  hipDeviceSynchronize();
  return 0;
}