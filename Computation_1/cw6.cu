
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 5
#define M 6
__global__ void init2DArray(unsigned *m) {
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  m[id] = id;
}
int main() {
  unsigned arr[N * M], *dmatrix;
  hipMalloc(&dmatrix, N * M * sizeof(unsigned));
  init2DArray<<<N, M>>>(dmatrix);
  hipMemcpy(arr, dmatrix, N * M * sizeof(int), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  for (int i = 0; i < N; i++)
    for (int j = 0; j < M; j++)
      printf("%d , ", arr[i * M + j]);
  printf("\n");
  return 0;
}